#include <omp.h>
#include <stdio.h>      // stdio functions are used since C++ streams aren't necessarily thread safe
#include <hipfft/hipfft.h> 
 
#define NX 1024
#define NY 1024
#define NZ 1024
 
using namespace std;

typedef unsigned char byte;


int fftmain()
{
        int num_gpus = 0;       // number of CUDA GPUs
 
	printf(" using CUDA and OpenMP \n");
        /////////////////////////////////////////////////////////////////
        // determine the number of CUDA capable GPUs
        //
    	hipGetDeviceCount(&num_gpus);
        if(num_gpus < 1)
        {
                printf("no CUDA capable devices were detected\n");
                return 1;
        }
 
        /////////////////////////////////////////////////////////////////
        // display CPU and GPU configuration
        //
    printf("number of host CPUs:\t%d\n", omp_get_num_procs());
    printf("number of CUDA devices:\t%d\n", num_gpus);
    for(int i = 0; i < num_gpus; i++)
    {
        hipDeviceProp_t dprop;
        hipGetDeviceProperties(&dprop, i);
                printf("   %d: %s\n", i, dprop.name);
    }
        printf("---------------------------\n");
 
 
     
 
    ////////////////////////////////////////////////////////////////
        // run as many CPU threads as there are CUDA devices
        //   each CPU thread controls a different device, processing its
        //   portion of the data.  It's possible to use more CPU threads
        //   than there are CUDA devices, in which case several CPU
        //   threads will be allocating resources and launching kernels
        //   on the same device.  For example, try omp_set_num_threads(2*num_gpus);
        //   Recall that all variables declared inside an "omp parallel" scope are
        //   local to each CPU thread
        //
        omp_set_num_threads(num_gpus);  // create as many CPU threads as there are CUDA devices
      //omp_set_num_threads(2*num_gpus);// create twice as many CPU threads as there are CUDA devices
#pragma omp parallel
    {
        unsigned int cpu_thread_id = omp_get_thread_num();
        unsigned int num_cpu_threads = omp_get_num_threads();
 
                // set and check the CUDA device for this CPU thread
                int gpu_id = -1;
                hipSetDevice(cpu_thread_id % num_gpus);        // "% num_gpus" allows more CPU threads than GPU devices
                hipGetDevice(&gpu_id);
 
                printf("CPU thread %d (of %d) uses CUDA device %d\n", cpu_thread_id, num_cpu_threads, gpu_id);

		// do the fft
		hipfftHandle plan;
		hipfftComplex *data1, *data2;
		hipMalloc((void**)&data1, sizeof( hipfftComplex)*NX*NY*NZ);
		hipMalloc((void**)&data2, sizeof( hipfftComplex)*NX*NY*NZ);
		/* Create a 3D FFT plan. */
		hipfftPlan3d(&plan, NX, NY, NZ, HIPFFT_C2C);

		/* Transform the first signal in place. */
		hipfftExecC2C(plan, data1, data1, HIPFFT_FORWARD);

		/* Transform the second signal using the same plan. */
		hipfftExecC2C(plan, data2, data2, HIPFFT_FORWARD);

		/* Destroy the cuFFT plan. */
		hipfftDestroy(plan);
		hipFree(data1); hipFree(data2);
  		
    }
        printf("---------------------------\n");
 
        if(hipSuccess != hipGetLastError())
                printf("%s\n", hipGetErrorString(hipGetLastError()));
 
    hipDeviceReset();
 
    return 0;
}



